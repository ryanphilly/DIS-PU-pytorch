#include "hip/hip_runtime.h"
//  Author: Yifan Wang yifan.wang@inf.ethz.ch
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include "cuda_utils.h"


__device__ void __update(float *__restrict__ dists, int *__restrict__ dists_i,
             int idx1, int idx2) {

    const float v1 = dists[idx1], v2 = dists[idx2];
    const int i1 = dists_i[idx1], i2 = dists_i[idx2];
    dists[idx1] = max(v1, v2);
    dists_i[idx1] = v2 > v1 ? i2 : i1;
}

// input: points(b, c, n) idx(b, m)
// output: out(b, c, m)
template <typename scalar_t>
__global__ void gather_points_forward_kernel(int b, int c, int n, int m,
                     const scalar_t *__restrict__ points,
                     const int *__restrict__ idx,
                     scalar_t *__restrict__ out) {
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int l = blockIdx.y; l < c; l += gridDim.y) {
            for (int j = threadIdx.x; j < m; j += blockDim.x) {
            int a = idx[i * m + j];
            out[(i * c + l) * m + j] = points[(i * c + l) * n + a];
            }
        }
    }
}

at::Tensor gather_points_cuda_forward(int b, int c, int n, int npoints,
                  at::Tensor points, at::Tensor idx,
                  at::Tensor out) {

    hipError_t err;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(points.type(), "gather_points_cuda_forward", ([&] {
            gather_points_forward_kernel<scalar_t><<<dim3(b, c, 1), opt_n_threads(npoints)>>>(
            b, c, n, npoints,
            points.data<scalar_t>(),
            idx.data<int32_t>(),
            out.data<scalar_t>());
        }));

    err = hipGetLastError();
    if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
    }
    return out;
}

// input: grad_out(b, c, m) idx(b, m)
// output: grad_points(b, c, n)
template <typename scalar_t>
__global__ void gather_points_backward_kernel(int b, int c, int n, int m,
                      scalar_t *__restrict__ grad_out,
                      const int *__restrict__ idx,
                      scalar_t *__restrict__ grad_points) {
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int l = blockIdx.y; l < c; l += gridDim.y) {
            for (int j = threadIdx.x; j < m; j += blockDim.x) {
            int a = idx[i * m + j];
            atomicAdd(grad_points + (i * c + l) * n + a,
                  grad_out[(i * c + l) * m + j]);
            }
        }
    }
}


at::Tensor gather_points_cuda_backward(int b, int c, int n, int npoints,
                       at::Tensor grad_out, at::Tensor idx, at::Tensor grad_points) {
    hipError_t err;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_out.type(), "gather_points_cuda_backward", ([&] {
        gather_points_backward_kernel<scalar_t><<<dim3(b, c, 1), opt_n_threads(npoints)>>>(
            b, c, n, npoints,
            grad_out.data<scalar_t>(),
            idx.data<int32_t>(),
            grad_points.data<scalar_t>());
      }));

    err = hipGetLastError();
    if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
    }
    return grad_points;
}


template <unsigned int block_size>
__global__ void furthest_point_sampling_forward_kernel(int b, int n, int m,
    const float * __restrict__ input, float * __restrict__ temp, int * __restrict__ idx) {
    // temp: (nxb) the closest distance from each of the n points to the existing set
    if (m <= 0) return;
    __shared__ float dists[block_size];
    __shared__ int dists_i[block_size];
    const unsigned int buffer_size = block_size;
    __shared__ float buf[block_size*3];
    for (int i=blockIdx.x; i<b; i+=gridDim.x){
        int old=0;
        // first out of sought m points is point0
        if (threadIdx.x==0) idx[i*m+0]=old;
        // fill buffer in the shared memory with input *once* for faster read
        for (int j=threadIdx.x;j<min(buffer_size,n)*3;j+=blockDim.x){
          buf[j]=input[i*n*3+j];
        }
        __syncthreads();
        // iteratively add m points
        for (int j=1; j<m; j++){
              int besti=0;
              float best=-1;
              // position of the last point
              float x1=input[i*n*3+old*3+0];
              float y1=input[i*n*3+old*3+1];
              float z1=input[i*n*3+old*3+2];
              // Neither do i understand this loop
              for (int k=threadIdx.x;k<n;k+=blockDim.x){
                float td=temp[blockIdx.x*n+k];
                float x2,y2,z2;
                // if buffer not filled, set new point an input point
                if (k<buffer_size){
                  x2=buf[k*3+0];
                  y2=buf[k*3+1];
                  z2=buf[k*3+2];
                }else{
                  x2=input[i*n*3+k*3+0];
                  y2=input[i*n*3+k*3+1];
                  z2=input[i*n*3+k*3+2];
                }
                float d=(x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
                float d2=min(d,td);
                if (d2!=td)
                  temp[blockIdx.x*n+k]=d2;
                if (d2>best){
                  best=d2;
                  besti=k;
                }
              }
              dists[threadIdx.x]=best;
              dists_i[threadIdx.x]=besti;
              // u from 0~log2(block_size)
              for (int u=0;(1<<u)<blockDim.x;u++){
                __syncthreads();
                // maximize pairwise between the current thread and
                // the sibling thread in the binary-tree
                if (threadIdx.x<(blockDim.x>>(u+1))){
                  int i1=(threadIdx.x*2)<<u;
                  int i2=(threadIdx.x*2+1)<<u;
                  if (dists[i1]<dists[i2]){
                    dists[i1]=dists[i2];
                    dists_i[i1]=dists_i[i2];
                  }
                }
              }
              __syncthreads();
              old=dists_i[0];
              if (threadIdx.x==0)
                idx[i*m+j]=old;
            }
          }
        }

at::Tensor furthest_sampling_cuda_forward(int b, int n, int m,
    at::Tensor input, at::Tensor temp, at::Tensor idx) {

    unsigned int n_threads = opt_n_threads(n);
    unsigned int n_blocks = min(32, (n*b + n_threads/2)/n_threads);
    switch (n_threads) {
      case 512:
      furthest_point_sampling_forward_kernel<512><<<n_blocks, n_threads>>>(
          b, n, m, input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 256:
      furthest_point_sampling_forward_kernel<256><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 128:
      furthest_point_sampling_forward_kernel<128><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 64:
      furthest_point_sampling_forward_kernel<64><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 32:
      furthest_point_sampling_forward_kernel<32><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 16:
      furthest_point_sampling_forward_kernel<16><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 8:
      furthest_point_sampling_forward_kernel<8><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 4:
      furthest_point_sampling_forward_kernel<4><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 2:
      furthest_point_sampling_forward_kernel<2><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      case 1:
      furthest_point_sampling_forward_kernel<1><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      break;
      default:
      furthest_point_sampling_forward_kernel<512><<<n_blocks, n_threads>>>(
          b, n, m,
          input.data<float>(),
          temp.data<float>(),
          idx.data<int32_t>());
      }

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
    }
    return idx;
}

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
template <typename scalar_t>
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int nsample,
                                        const scalar_t *__restrict__ new_xyz,
                                        const scalar_t *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    scalar_t new_x = new_xyz[j * 3 + 0];
    scalar_t new_y = new_xyz[j * 3 + 1];
    scalar_t new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      scalar_t x = xyz[k * 3 + 0];
      scalar_t y = xyz[k * 3 + 1];
      scalar_t z = xyz[k * 3 + 2];
      scalar_t d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

at::Tensor ball_query_cuda_forward(int b, int n, int m, float radius,
                                     int nsample, at::Tensor query,
                                     at::Tensor xyz, at::Tensor idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(xyz.type(), "query_ball_point_kernel", ([&]() {
    query_ball_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(b, n, m, radius, nsample, 
      query.data<scalar_t>(), xyz.data<scalar_t>(), idx.data<int32_t>());
		  }));
  CUDA_CHECK_ERRORS();
  return idx;
}